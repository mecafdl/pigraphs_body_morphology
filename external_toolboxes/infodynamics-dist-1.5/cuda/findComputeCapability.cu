
#include <hip/hip_runtime.h>
#include <cstdio>
int main() {
  int count = 0;
  if (hipSuccess != hipGetDeviceCount(&count)) return -1;
  if (count == 0) return -1;
  if (count == 1) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::printf("-arch=sm_%d%d", prop.major, prop.minor);
  } else {
    for (int device = 0; device < count; ++device) {
      hipDeviceProp_t prop;
      if (hipSuccess == hipGetDeviceProperties(&prop, device)) {
        std::printf("-gencode arch=compute_%d%d,code=sm_%d%d ", prop.major, prop.minor, prop.major, prop.minor);
      }
    }
  }
  return 0;
}
